#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<sys/time.h>
#include <assert.h>

#include <string>
#include"ppi-bpt.h"
#include"cuda_utils.h"
#include"mempool.h"

#include <fstream>
#include <omp.h>
#include "hipcub/hipcub.hpp"
#define M 1000000
#define Keys_Count 100*M

#define GPU_SWP_SIZE  16
#define PPI_Block_Dim 128
#define PPI_Block_Per_SM 64
#define PPI_Thread_Per_Query 8
#define  Sort_Per_Thread (GPU_SWP_SIZE / PPI_Thread_Per_Query)

#define GPU_SWP_SIZE4  16
#define PPI_Block_Dim_4thread 128
#define PPI_Block_Per_SM_4thread 64
#define PPI_Thread_Per_Query_4thread 4
#define  Sort_Per_4Thread (GPU_SWP_SIZE4 / PPI_Thread_Per_Query_4thread)

typedef struct{
    int blfnode;
    //BLeaf_node *blfnode;
    char relist_idx;
}GPU_Result;

using namespace std;

/*--------------------------------------------------------------------------
*
*   
*           PPI_BPT_V2
*           using 8 thread
*           sort first
*           batch 
*           key pos back
*           whole
*
*-----------------------------------------------------------------------------*/

__global__ void ppi_bpt_V2_search_kernel_8threads(Inner_node *d_innode,  int root_idx,  key_t *d_keys, int tree_height, GPU_Result *d_gresult, int key_count){
    /*{{{*/ 
    int key_base = PPI_Block_Dim / PPI_Thread_Per_Query * GPU_SWP_SIZE ;
    int key_idx = key_base * blockIdx.x + threadIdx.x/ PPI_Thread_Per_Query;  

    if(key_idx>=key_count) return;

    int search_idx = threadIdx.x % PPI_Thread_Per_Query;
    int query_idx_in_local_block = threadIdx.x / PPI_Thread_Per_Query;
    const int r = query_idx_in_local_block;//just for simple

    const int row = PPI_Block_Dim/ PPI_Thread_Per_Query;//blockDim.x/ PPI_Thread_Per_Query;
    const int row_swp = row * GPU_SWP_SIZE ;
    __shared__ char flag[row][9];
    __shared__ int inner_index_result[row];
    __shared__ int start_idx[row_swp];

    int stride = PPI_Block_Dim/ PPI_Thread_Per_Query;
    
    /*
    for(int i=0;i<GPU_SWP_SIZE2;i++){
        int cur_key = key_idx+stride*i;
        if(cur_key>=key_count)continue;
        start_idx[r + stride*i] = d_inter_result[cur_key];
    }
*/
 
    for (int k = 0; k<Sort_Per_Thread; k++){
        start_idx[threadIdx.x + k* blockDim.x] = root_idx;
    }

    flag[r][search_idx] = 0;
    flag[r][search_idx+1] = 0;

    char selfFlag;
    __syncthreads();

    key_t target;

    for(int i=1;i<tree_height;i++){
        for(int j=0;j<GPU_SWP_SIZE ;j++){
            __syncthreads();
            int cur_key = key_idx+stride *j;
            if(cur_key>=key_count)continue;

            int cur_r = r+stride *j;

            target = d_keys[cur_key];
            GPU_Result &result=d_gresult[cur_key];

            int pos = start_idx[cur_r];
            Inner_node *node = d_innode + pos;
            //search index;
            key_t idx_key = node->inner_index[search_idx];
            
            if(target < idx_key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
            }
            __syncthreads();
            
           
            if(selfFlag == 1 && flag[r][search_idx] == 0){
                inner_index_result[r] = search_idx; 
            }
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            __syncthreads();
    //-----------------------------------------------------------------------------------------
            //search key;
            int begin = inner_index_result[r]*8;
            int idx = begin + search_idx;
            key_t key = node->inner_key[idx];

            //===== shit
            if(search_idx == 0){
                if(i == tree_height-1){
                    result.blfnode = (int)(BLeaf_node *)node->child[0];
                    result.relist_idx = begin+8;
                }else
                    start_idx[cur_r] = (long)node->child[begin+8];
            }
            //=====

            if(target < key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
            }
            __syncthreads();
            
            //get next child;
            if(selfFlag == 1 && flag[r][search_idx] == 0){
                if(i==tree_height-1){
                    result.relist_idx = idx ;
                    //return;
                }else
                    start_idx[cur_r] = (long)node->child[idx];
            }
            inner_index_result[r] = 0;
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            __syncthreads();
        
        }
    }/*}}}*/
}

void PPI_BPT_Search_GPU_V2_8thread(BPlusTree &tree,ifstream &search_file, int startBit, int endBit){
/*{{{*/
    Inner_node *d_innode = prepareGPU_v1(tree);

    int rootIdx = tree.getRootIdx();
    
    
    assert(rootIdx != -1);
    int Thread_Per_Block = PPI_Block_Dim;
    int Block_Per_Grid = 13 * PPI_Block_Per_SM ;
    int Para_Search_Bucket = (13 * PPI_Block_Per_SM * Thread_Per_Block) / PPI_Thread_Per_Query*GPU_SWP_SIZE ;
    
    dim3 block_dim(Thread_Per_Block);
    dim3 grid_dim(Block_Per_Grid);

    int batch_d_key_size = sizeof(key_t) * Para_Search_Bucket;
    int batch_gresult_size = sizeof(GPU_Result) * Para_Search_Bucket;
    int batch_pos_size = sizeof(int) * Para_Search_Bucket;
    
    //host_malloc
    //key_t host_keys[Keys_Count];
    key_t *host_keys = (key_t *)malloc(Keys_Count * sizeof(key_t));
    int *host_pos = (int *)malloc(batch_pos_size);

    GPU_Result  *h_gresult = (GPU_Result *)malloc(batch_gresult_size);
    string s;
    int nums = 0;

    while(getline(search_file,s)) {
        key_t key;
        sscanf(s.c_str(),TYPE_D,&key);
        host_keys[nums++] = key;
        //cout<<key<<endl;
    }
     
    for (int i=0;i<Para_Search_Bucket;i++) {
        host_pos[i] = i;
    }


    //gpu_malloc
    key_t *d_keys;
    CUDA_ERROR_HANDLER(hipMalloc(&d_keys, batch_d_key_size));
    

    key_t *d_keys_after;
    CUDA_ERROR_HANDLER(hipMalloc(&d_keys_after, batch_d_key_size));
  
    int *d_pos;
    CUDA_ERROR_HANDLER(hipMalloc(&d_pos, batch_pos_size));

    int *d_pos_after;
    CUDA_ERROR_HANDLER(hipMalloc(&d_pos_after, batch_pos_size));

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    GPU_Result *d_gresult;
    CUDA_ERROR_HANDLER(hipMalloc(&d_gresult, batch_gresult_size ));


    
    int kernel_height = tree.getHeight() ;
    int total = nums / Para_Search_Bucket;


    value_t val[Para_Search_Bucket];
   


    struct timeval start;
    struct timeval end;
    double t_gpu_transfer_1 = 0;
    double t_gpu_sort = 0;
    double t_gpu_whole = 0;
    double t_gpu_transfer_2 = 0;
    double t_cpu=0;

    CUDA_ERROR_HANDLER(hipMemcpy(d_pos, host_pos, batch_pos_size, hipMemcpyHostToDevice));

    float time_gpu = 0;

    hipEvent_t g_start;
    hipEvent_t g_stop;
        
    hipEventCreate(&g_start);
    hipEventCreate(&g_stop);
    for (int i=0;i<total;i++) {

        
        
        hipEventRecord(g_start);
        CUDA_ERROR_HANDLER(hipMemcpy(d_keys, host_keys + i * Para_Search_Bucket, batch_d_key_size, hipMemcpyHostToDevice));
        hipEventRecord(g_stop);
        hipEventSynchronize(g_stop);
        
        hipEventElapsedTime(&time_gpu,g_start,g_stop);
        

        t_gpu_transfer_1 += time_gpu/1000;
        //t_gpu_transfer_1 += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------
        
        if (i==0) {
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos,d_pos_after,Para_Search_Bucket, startBit, endBit);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
        }

        hipEventRecord(g_start);
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos, d_pos_after, Para_Search_Bucket,startBit,endBit);
        hipEventRecord(g_stop);
        hipEventSynchronize(g_stop);

        hipEventElapsedTime(&time_gpu,g_start,g_stop);


        t_gpu_sort += time_gpu/1000;
        //t_gpu_sort += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------

        
        hipEventRecord(g_start);
        ppi_bpt_V2_search_kernel_8threads<<<grid_dim, block_dim>>>(d_innode, rootIdx , d_keys_after, kernel_height, d_gresult, Para_Search_Bucket);
        hipEventRecord(g_stop);
        hipEventSynchronize(g_stop);
        
        hipEventElapsedTime(&time_gpu,g_start,g_stop);

        t_gpu_whole += time_gpu/1000;
        //t_gpu_whole += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;

//---------------------------------------------------------------------

        hipEventRecord(g_start);
        CUDA_ERROR_HANDLER(hipMemcpy(h_gresult, d_gresult, batch_gresult_size, hipMemcpyDeviceToHost));
         
        //CUDA_ERROR_HANDLER(hipMemcpy(host_keys_after, d_keys_after, batch_d_key_size, hipMemcpyDeviceToHost));
        CUDA_ERROR_HANDLER(hipMemcpy(host_pos, d_pos_after, batch_pos_size, hipMemcpyDeviceToHost));
        
        hipEventRecord(g_stop);
        hipEventSynchronize(g_stop);
        
        hipEventElapsedTime(&time_gpu,g_start,g_stop);

        t_gpu_transfer_2 += time_gpu / 1000;
        //t_gpu_transfer_2 += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------------------
        int ss = i * Para_Search_Bucket;

        gettimeofday(&start, NULL); 
       #pragma omp parallel for
        for (int j=0; j<Para_Search_Bucket;j++) {
            key_t key = host_keys[host_pos[j]+ss];
            BLeaf_node *blfnode = tree.getLeafByIdx(h_gresult[j].blfnode);
            val[j] = blfnode->findKey(h_gresult[j].relist_idx, key);
        }
    
        gettimeofday(&end, NULL);
        t_cpu += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
        //test
        //for (int j=0;j<Para_Search_Bucket;j++) {
        //     key_t key = host_keys[host_pos[j]+ss];
        //
        //    cout<<key<<": "<<val[j]<<endl;
        //}
    }

    //cout<<"GPU PPI V3 [sort key first, batch, 8threads,whole, pos_back, blfnode_int]"<<total * Para_Search_Bucket<<endl;
    //cout<<"GPU search total num:"<<total * Para_Search_Bucket<<endl;
    cout<<"t_gpu_transfer_1:    "<<t_gpu_transfer_1<<endl;
    cout<<"t_gpu_sort:          "<<t_gpu_sort<<endl;
    cout<<"t_gpu_whole:         "<<t_gpu_whole<<endl;
    cout<<"t_gpu_transfer_2:    "<<t_gpu_transfer_2<<endl;
    cout<<"t_cpu:               "<<t_cpu<<endl;
    cout<<"t_gpu:               "<<t_gpu_whole+ t_gpu_sort << endl;
    cout<<"total time:          "<<t_gpu_whole +t_gpu_transfer_1 +t_gpu_transfer_2+  t_gpu_sort + t_cpu<<endl;;
    
    
    CUDA_ERROR_HANDLER(hipFree(d_innode));
    CUDA_ERROR_HANDLER(hipFree(d_keys));
    CUDA_ERROR_HANDLER(hipFree(d_keys_after));
    CUDA_ERROR_HANDLER(hipFree(d_temp_storage));
    CUDA_ERROR_HANDLER(hipFree(d_gresult));

/*}}}*/

}
void PPI_BPT_Search_GPU_V2_8thread(BPlusTree &tree, ifstream &search_file){
    PPI_BPT_Search_GPU_V2_8thread(tree, search_file, 48, 64);
}

/*--------------------------------------------------------------------------
*
*   
*           PPI_BPT_V2
*           using 2 thread
*           batch
*           key pos sort
*           whole
*
*-----------------------------------------------------------------------------*/



__global__ void ppi_bpt_V2_search_kernel_4threads(Inner_node *d_innode,  int root_idx,  key_t *d_keys, int tree_height, GPU_Result *d_gresult, int key_count){
    /*{{{*/ 
    int key_base = PPI_Block_Dim_4thread / PPI_Thread_Per_Query_4thread * GPU_SWP_SIZE4;
    int key_idx = key_base * blockIdx.x + threadIdx.x/ PPI_Thread_Per_Query_4thread;  

    if(key_idx>=key_count) return;

    int search_idx = threadIdx.x % PPI_Thread_Per_Query_4thread;
    int query_idx_in_local_block = threadIdx.x / PPI_Thread_Per_Query_4thread;
    const int r = query_idx_in_local_block;//just for simple

    const int row = PPI_Block_Dim_4thread/ PPI_Thread_Per_Query_4thread;//blockDim.x/ PPI_Thread_Per_Query;
    const int row_swp = row * GPU_SWP_SIZE4;
    __shared__ char flag[row][5];
    __shared__ int inner_index_result[row];
    __shared__ int start_idx[row_swp];

    __shared__ char nexthalf[row];

    int stride = PPI_Block_Dim_4thread / PPI_Thread_Per_Query_4thread;
    
   
    for (int k = 0; k<Sort_Per_4Thread; k++){
        start_idx[threadIdx.x + k* blockDim.x] = root_idx;
    }

    flag[r][search_idx] = 0;
    flag[r][search_idx+1] = 0;

    char selfFlag;
    __syncthreads();

    key_t target;

    for(int i=1;i<tree_height;i++){
        for(int j=0;j<GPU_SWP_SIZE4;j++){
            nexthalf[r] = -1;
            __syncthreads();
            int cur_key = key_idx+stride *j;
            if(cur_key>=key_count)continue;

            int cur_r = r+stride *j;

            target = d_keys[cur_key];
            GPU_Result &result=d_gresult[cur_key];

            int pos = start_idx[cur_r];
            Inner_node *node = d_innode + pos;
            //search index;
            key_t idx_key = node->inner_index[search_idx];
            
            if(target < idx_key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
                nexthalf[r] = 0;
            }
            __syncthreads();
            
            if(nexthalf[r] == -1){
                idx_key = node->inner_index[search_idx+4];
                if(target < idx_key){
                    flag[r][search_idx+1] = 1;
                    selfFlag = 1;
                    nexthalf[r] = 1;
                }
                __syncthreads();
            }

            if(selfFlag == 1 && flag[r][search_idx] == 0){
                inner_index_result[r] = search_idx+nexthalf[r]*4; 
            }
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            nexthalf[r] = -1;
            __syncthreads();
    //-----------------------------------------------------------------------------------------
            //search key;
            int begin = inner_index_result[r]*8;
            int idx = begin + search_idx;
            key_t key = node->inner_key[idx];

            //===== shit
            if(search_idx == 0){
                if(i == tree_height-1){
                    result.blfnode = (int)(BLeaf_node *)node->child[0];
                    result.relist_idx = begin+8;
                }else
                    start_idx[cur_r] = (long)node->child[begin+8];
            }
            //=====

            if(target < key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
                nexthalf[r] = 0;
            }
            __syncthreads();
            if(nexthalf[r] == -1){
                key = node->inner_key[idx+4];
                if(target < key){
                    flag[r][search_idx+1] = 1;
                    selfFlag = 1;
                    nexthalf[r] = 1;
                }
                __syncthreads();
            }
     
            
            //get next child;
            if(selfFlag == 1 && flag[r][search_idx] == 0){
                if(i==tree_height-1){

                    result.relist_idx = idx + nexthalf[r]*4;
                    //return;
                }else
                    start_idx[cur_r] = (long)node->child[idx+nexthalf[r]*4];
            }
            inner_index_result[r] = 0;
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            __syncthreads();
        
        }
    }/*}}}*/
}
void PPI_BPT_Search_GPU_V2_4thread(BPlusTree &tree,ifstream &search_file){
/*{{{*/
    Inner_node *d_innode = prepareGPU_v1(tree);

    int rootIdx = tree.getRootIdx();
    
    
    assert(rootIdx != -1);
    int Thread_Per_Block = PPI_Block_Dim_4thread;
    int Block_Per_Grid = 13 * PPI_Block_Per_SM_4thread ;
    int Para_Search_Bucket = (13 * PPI_Block_Per_SM_4thread * Thread_Per_Block) / PPI_Thread_Per_Query_4thread *GPU_SWP_SIZE4;
    
    dim3 block_dim(Thread_Per_Block);
    dim3 grid_dim(Block_Per_Grid);

    int batch_d_key_size = sizeof(key_t) * Para_Search_Bucket;
    int batch_gresult_size = sizeof(GPU_Result) * Para_Search_Bucket;
    int batch_pos_size = sizeof(int) * Para_Search_Bucket;
    
    
    //host_malloc
    //key_t host_keys[Keys_Count];
    key_t *host_keys = (key_t *)malloc(Keys_Count * sizeof(key_t));
    int *host_pos = (int *)malloc(batch_pos_size);

    GPU_Result  *h_gresult = (GPU_Result *)malloc(batch_gresult_size);
    string s;
    int nums = 0;

    while(getline(search_file,s)) {
        key_t key;
        sscanf(s.c_str(),TYPE_D,&key);
        host_keys[nums++] = key;
        //cout<<key<<endl;
    }
     
    for (int i=0;i<Para_Search_Bucket;i++) {
        host_pos[i] = i;
    }


    //gpu_malloc
    key_t *d_keys;
    CUDA_ERROR_HANDLER(hipMalloc(&d_keys, batch_d_key_size));
    

    key_t *d_keys_after;
    CUDA_ERROR_HANDLER(hipMalloc(&d_keys_after, batch_d_key_size));
  
    int *d_pos;
    CUDA_ERROR_HANDLER(hipMalloc(&d_pos, batch_pos_size));

    int *d_pos_after;
    CUDA_ERROR_HANDLER(hipMalloc(&d_pos_after, batch_pos_size));

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    GPU_Result *d_gresult;
    CUDA_ERROR_HANDLER(hipMalloc(&d_gresult, batch_gresult_size ));


    
    int kernel_height = tree.getHeight() ;
    int total = nums / Para_Search_Bucket;


    value_t val[Para_Search_Bucket];
   


    struct timeval start;
    struct timeval end;
    double t_gpu_transfer_1 = 0;
    double t_gpu_sort = 0;
    double t_gpu_whole = 0;
    double t_gpu_transfer_2 = 0;
    double t_cpu=0;

    CUDA_ERROR_HANDLER(hipMemcpy(d_pos, host_pos, batch_pos_size, hipMemcpyHostToDevice));


    for (int i=0;i<total;i++) {

        gettimeofday(&start, NULL); 
        
        CUDA_ERROR_HANDLER(hipMemcpy(d_keys, host_keys + i * Para_Search_Bucket, batch_d_key_size, hipMemcpyHostToDevice));
        
        gettimeofday(&end, NULL); 
        t_gpu_transfer_1 += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------
       gettimeofday(&start, NULL); 
        
        if (i==0) {
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos,d_pos_after,Para_Search_Bucket);
            hipMalloc(&d_temp_storage, temp_storage_bytes);
        }
#ifdef TREE_32
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos, d_pos_after, Para_Search_Bucket,16,32);
#else
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos, d_pos_after, Para_Search_Bucket,48,64);
#endif
        hipDeviceSynchronize();

        gettimeofday(&end, NULL);

        t_gpu_sort += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------

        gettimeofday(&start, NULL); 
        
        ppi_bpt_V2_search_kernel_4threads<<<grid_dim, block_dim>>>(d_innode, rootIdx , d_keys_after, kernel_height, d_gresult, Para_Search_Bucket);

        hipDeviceSynchronize();
        gettimeofday(&end, NULL);
        t_gpu_whole += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;

        gettimeofday(&start, NULL); 
//---------------------------------------------------------------------

        CUDA_ERROR_HANDLER(hipMemcpy(h_gresult, d_gresult, batch_gresult_size, hipMemcpyDeviceToHost));
         
        //CUDA_ERROR_HANDLER(hipMemcpy(host_keys_after, d_keys_after, batch_d_key_size, hipMemcpyDeviceToHost));
        CUDA_ERROR_HANDLER(hipMemcpy(host_pos, d_pos_after, batch_pos_size, hipMemcpyDeviceToHost));

        gettimeofday(&end, NULL);

        t_gpu_transfer_2 += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
//---------------------------------------------------------------------------------
        int ss = i * Para_Search_Bucket;

        gettimeofday(&start, NULL); 
        #pragma omp parallel for
        for (int j=0; j<Para_Search_Bucket;j++) {
            key_t key = host_keys[host_pos[j]+ss];
            BLeaf_node *blfnode = tree.getLeafByIdx(h_gresult[j].blfnode);
            val[j] = blfnode->findKey(h_gresult[j].relist_idx, key);
        }
    
        gettimeofday(&end, NULL);
        t_cpu += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;
        //test
        //for (int j=0;j<Para_Search_Bucket;j++) {
        //     key_t key = host_keys[host_pos[j]+ss];
        //
        //    cout<<key<<": "<<val[j]<<endl;
        //}
    }

    cout<<"GPU PPI V3 [sort key first, batch, 4threads,whole, pos_back, blfnode_int]"<<total * Para_Search_Bucket<<endl;
    cout<<"GPU search total num:"<<total * Para_Search_Bucket<<endl;
    cout<<"t_gpu_transfer_1:    "<<t_gpu_transfer_1<<endl;
    cout<<"t_gpu_sort:          "<<t_gpu_sort<<endl;
    cout<<"t_gpu_whole:         "<<t_gpu_whole<<endl;
    cout<<"t_gpu_transfer_2:    "<<t_gpu_transfer_2<<endl;
    cout<<"t_cpu:               "<<t_cpu<<endl;
    cout<<"t_gpu:               "<<t_gpu_whole+ t_gpu_sort << endl;
    cout<<"total time:          "<<t_gpu_whole +t_gpu_transfer_1 +t_gpu_transfer_2+  t_gpu_sort + t_cpu<<endl;;
    
    
    CUDA_ERROR_HANDLER(hipFree(d_innode));
    CUDA_ERROR_HANDLER(hipFree(d_keys));
    CUDA_ERROR_HANDLER(hipFree(d_keys_after));
    CUDA_ERROR_HANDLER(hipFree(d_temp_storage));
    CUDA_ERROR_HANDLER(hipFree(d_gresult));

/*}}}*/

}


