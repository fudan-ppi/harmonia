#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<sys/time.h>
#include <assert.h>

#include <string>
#include"ppi-bpt.h"
#include"cuda_utils.h"
#include"mempool.h"
#include<sched.h>
#include<pthread.h>
#include<unistd.h>

#include <fstream>
#include <omp.h>
#include "hipcub/hipcub.hpp"

#define M 1000000
#define Keys_Count 100*M

#define  GPU_SWP_SIZE2 16 


#define PPI_Block_Dim_2thread 128
#define PPI_Block_Per_SM 64

#define PPI_Thread_Per_Query_2thread 2
#define  Sort_Per_2Thread (GPU_SWP_SIZE2 / PPI_Thread_Per_Query_2thread)

#define CPU_THREAD 54
#define UP_THREAD 30
#define DOWN_THREAD 24
#define UP_HEIGHT 2


typedef struct{
    int blfnode;
    char relist_idx;
}GPU_Result;

using namespace std;





__global__ void ppi_bpt_V6_search_kernel_2threads_new_tree_balance(Inner_node *d_innode, int* d_prefix, int inner_node_size_wo_last_inner_node, int* d_inter_result, int* d_pos,  key_t *d_keys, int tree_height, GPU_Result *d_gresult, int key_count){
    /*{{{*/ 
    int key_base = PPI_Block_Dim_2thread / PPI_Thread_Per_Query_2thread * GPU_SWP_SIZE2;
    int key_idx = key_base * blockIdx.x + threadIdx.x/ PPI_Thread_Per_Query_2thread;  

    if(key_idx>=key_count) return;

    int search_idx = threadIdx.x % PPI_Thread_Per_Query_2thread;
    int query_idx_in_local_block = threadIdx.x / PPI_Thread_Per_Query_2thread;
    const int r = query_idx_in_local_block;//just for simple

    const int row = PPI_Block_Dim_2thread/ PPI_Thread_Per_Query_2thread;//blockDim.x/ PPI_Thread_Per_Query;
    const int row_swp = row * GPU_SWP_SIZE2;
    __shared__ char flag[row][3];
    __shared__ int inner_index_result[row];
    __shared__ int start_idx[row_swp];

    __shared__ char nexthalf[row];

    int stride = PPI_Block_Dim_2thread / PPI_Thread_Per_Query_2thread;
    
    for (int i=0; i<Sort_Per_2Thread;i++) {
       int cur_key = key_base * blockIdx.x + i*blockDim.x + threadIdx.x;
       //if (cur_key>=key_count) break;
       start_idx[threadIdx.x + i*blockDim.x] = d_inter_result[d_pos[cur_key]];
    }

    flag[r][search_idx] = 0;
    flag[r][search_idx+1] = 0;

    char selfFlag;
    __syncthreads();

    key_t target;

    for(int i=1;i<tree_height;i++){
        for(int j=0;j<GPU_SWP_SIZE2;j++){
            nexthalf[r] = -1;
            __syncthreads();
            int cur_key = key_idx+stride *j;
            if(cur_key>=key_count)continue;

            int cur_r = r+stride *j;

            target = d_keys[cur_key];
            GPU_Result &result=d_gresult[cur_key];

            int pos = start_idx[cur_r];
            Inner_node *node = d_innode + pos;
            //search index;
            key_t idx_key = node->inner_index[search_idx];
            
            if(target < idx_key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
                nexthalf[r] = 0;
            }
            __syncthreads();
            
            if(nexthalf[r] == -1){
                idx_key = node->inner_index[search_idx+2];
                if(target < idx_key){
                    flag[r][search_idx+1] = 1;
                    selfFlag = 1;
                    nexthalf[r] = 1;
                }
                __syncthreads();

                if(nexthalf[r] == -1){
                    idx_key = node->inner_index[search_idx+4];
                    if(target < idx_key){
                        flag[r][search_idx+1] = 1;
                        selfFlag = 1;
                        nexthalf[r] = 2;
                    }
                    __syncthreads();
                    if(nexthalf[r] == -1){
                        idx_key = node->inner_index[search_idx+6];
                        if(target < idx_key){
                            flag[r][search_idx+1] = 1;
                            selfFlag = 1;
                            nexthalf[r] = 3;
                        }
                        __syncthreads();
                    }
                }
            }

            if(selfFlag == 1 && flag[r][search_idx] == 0){
                inner_index_result[r] = search_idx+nexthalf[r]*2; 
            }
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            nexthalf[r] = -1;
            __syncthreads();
    //-----------------------------------------------------------------------------------------
            //search key;
            int begin = inner_index_result[r]*8;
            int idx = begin + search_idx;
            key_t key = node->inner_key[idx];

            //===== shit
            if(search_idx == 0){
                if(i == tree_height-1){
                    
                    //result.blfnode = (BLeaf_node *)node->child[0];
                    result.blfnode = pos - inner_node_size_wo_last_inner_node;
                    result.relist_idx = begin+8;
                }else
                    start_idx[cur_r] = __ldg(&d_prefix[pos]) + (begin+8);
            }
            //=====

            if(target < key){
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
                nexthalf[r] = 0;
            }
            __syncthreads();
            if(nexthalf[r] == -1){
                key = node->inner_key[idx+2];
                if(target < key){
                    flag[r][search_idx+1] = 1;
                    selfFlag = 1;
                    nexthalf[r] = 1;
                }
                __syncthreads();

                if(nexthalf[r] == -1){
                    key = node->inner_key[idx+4];
                    if(target < key){
                        flag[r][search_idx+1] = 1;
                        selfFlag = 1;
                        nexthalf[r] = 2;
                    }
                    __syncthreads();
                    if(nexthalf[r] == -1){
                        key = node->inner_key[idx+6];
                        if(target < key){
                            flag[r][search_idx+1] = 1;
                            selfFlag = 1;
                            nexthalf[r] = 3;
                        }
                        __syncthreads();
                    }
                }
            }

     
            
            //get next child;
            if(selfFlag == 1 && flag[r][search_idx] == 0){
                if(i==tree_height-1){

                    //result.blfnode = pos - inner_node_size_wo_last_inner_node;
                    result.relist_idx = idx + nexthalf[r]*2;
                    //return;
                }else 
                    start_idx[cur_r] = __ldg(&d_prefix[pos]) + (idx + nexthalf[r]*2);
            }
            inner_index_result[r] = 0;
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            __syncthreads();
        
        }
    }/*}}}*/
}





namespace{
    volatile int stream_flag[2] = {-1,-1};/*{{{*/
    int Thread_Per_Block = PPI_Block_Dim_2thread;
    int Block_Per_Grid = 13 * PPI_Block_Per_SM ;
    const int Para_Search_Bucket = (Block_Per_Grid * Thread_Per_Block) / PPI_Thread_Per_Query_2thread * GPU_SWP_SIZE2;

    dim3 block_dim(Thread_Per_Block);
    dim3 grid_dim(Block_Per_Grid);

    int batch_d_key_size = sizeof(key_t) * Para_Search_Bucket;
    int batch_gresult_size = sizeof(GPU_Result) * Para_Search_Bucket;
    int batch_pos_size = sizeof(int) * Para_Search_Bucket;
    int batch_inter_size = sizeof(int) * Para_Search_Bucket;

    key_t *host_keys;
    int *host_pos; 
    GPU_Result  *h_gresult;
    int *host_inter_result;

    key_t *d_keys;
    key_t *d_keys_after;
    int *d_pos;
    int *d_pos_after;
    GPU_Result *d_gresult;
    int *d_inter_result;

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    int rootIdx; 
    Inner_node *inner_start;
    Inner_node *root;

    int kernel_height;
    int up_height;
    int inner_node_size_wo_last_inner_node;
    int total;

    Inner_node *d_innode;
    int *d_prefix;
    hipStream_t stream[2];


    void CUDART_CB CallBack(hipStream_t stream,hipError_t status,void *data){
        stream_flag[(size_t)data%2] = (size_t)data;
    }
/*}}}*/
    float time_gpu = 0;

    hipEvent_t g_start,g_stop;

    //
    hipEvent_t *g_starts;
    hipEvent_t *g_stops;
    //
    
    BPlusTree *bptree;
    vector<double> cpu_thread_total_time(DOWN_THREAD,0);
    vector<double> cpu_thread_compute_time(DOWN_THREAD,0);


    volatile int key_status = -1;
}



void* launch_kernel_thread_new_tree_measure_balance(void *args){
/*{{{*/
    stick_this_thread_to_core(1);
    hipEventCreate(&g_start);
    hipEventCreate(&g_stop);
    


    hipEventRecord(g_start);
    for(int i=0;i<total;i++){
        while (key_status<i);
        int idx = i%2;
        int stride = idx*Para_Search_Bucket;
        const key_t *tmp_dkeys =d_keys+stride;
        const int *tmp_dpos =d_pos+stride;
        

        CUDA_ERROR_HANDLER(hipMemcpyAsync(d_keys + stride, host_keys + i * Para_Search_Bucket, batch_d_key_size, hipMemcpyHostToDevice,stream[idx]));
        CUDA_ERROR_HANDLER(hipMemcpyAsync(d_inter_result + stride, host_inter_result + i * Para_Search_Bucket, batch_inter_size, hipMemcpyHostToDevice, stream[idx]));

#ifdef TREE_32
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage + idx*temp_storage_bytes, temp_storage_bytes, tmp_dkeys, d_keys_after+stride, tmp_dpos, d_pos_after+stride, Para_Search_Bucket,16,32,stream[idx]);

#else 
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage + idx*temp_storage_bytes, temp_storage_bytes, tmp_dkeys, d_keys_after+stride, tmp_dpos, d_pos_after+stride, Para_Search_Bucket,48,64,stream[idx]);
#endif  

        ppi_bpt_V6_search_kernel_2threads_new_tree_balance<<<grid_dim, block_dim,0,stream[idx]>>>(d_innode,d_prefix, inner_node_size_wo_last_inner_node, d_inter_result+stride, d_pos_after+stride, d_keys_after+stride, kernel_height , d_gresult+stride, Para_Search_Bucket);
        CUDA_ERROR_HANDLER(hipMemcpyAsync(h_gresult+i*Para_Search_Bucket, d_gresult+stride, batch_gresult_size, hipMemcpyDeviceToHost,stream[idx]));
        CUDA_ERROR_HANDLER(hipMemcpyAsync(host_pos+i*Para_Search_Bucket, d_pos_after+stride, batch_pos_size, hipMemcpyDeviceToHost,stream[idx]));

        hipStreamAddCallback(stream[idx],CallBack,(void *)i,0);


    }
    
    hipEventRecord(g_stop);
   
    return NULL;/*}}}*/
}

void* launch_kernel_thread_new_tree_measure2_balance(void *args){
/*{{{*/
    stick_this_thread_to_core(1);

    g_starts = (hipEvent_t *)malloc(sizeof(hipEvent_t)*total);
    g_stops = (hipEvent_t *)malloc(sizeof(hipEvent_t)*total);
    for (int i=0;i<total;i++) {
   
        hipEventCreate(&g_starts[i]);
        hipEventCreate(&g_stops[i]);
    }
    


    //hipEventRecord(g_start);
    for(int i=0;i<total;i++){
        while (key_status<i);
        int idx = i%2;
        int stride = idx*Para_Search_Bucket;
        const key_t *tmp_dkeys =d_keys+stride;
        const int *tmp_dpos =d_pos+stride;
       
        hipEventRecord(g_starts[i], stream[idx]);

        CUDA_ERROR_HANDLER(hipMemcpyAsync(d_keys + stride, host_keys + i * Para_Search_Bucket, batch_d_key_size, hipMemcpyHostToDevice,stream[idx]));
        CUDA_ERROR_HANDLER(hipMemcpyAsync(d_inter_result + stride, host_inter_result + i * Para_Search_Bucket, batch_inter_size, hipMemcpyHostToDevice, stream[idx]));

#ifdef TREE_32
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage + idx*temp_storage_bytes, temp_storage_bytes, tmp_dkeys, d_keys_after+stride, tmp_dpos, d_pos_after+stride, Para_Search_Bucket,16,32,stream[idx]);
#else 

        hipcub::DeviceRadixSort::SortPairs(d_temp_storage + idx*temp_storage_bytes, temp_storage_bytes, tmp_dkeys, d_keys_after+stride, tmp_dpos, d_pos_after+stride, Para_Search_Bucket,48,64,stream[idx]);
#endif  

        ppi_bpt_V6_search_kernel_2threads_new_tree_balance<<<grid_dim, block_dim,0,stream[idx]>>>(d_innode,d_prefix, inner_node_size_wo_last_inner_node, d_inter_result+stride, d_pos_after+stride, d_keys_after+stride, kernel_height , d_gresult+stride, Para_Search_Bucket);
        CUDA_ERROR_HANDLER(hipMemcpyAsync(h_gresult+i*Para_Search_Bucket, d_gresult+stride, batch_gresult_size, hipMemcpyDeviceToHost,stream[idx]));
        CUDA_ERROR_HANDLER(hipMemcpyAsync(host_pos+i*Para_Search_Bucket, d_pos_after+stride, batch_pos_size, hipMemcpyDeviceToHost,stream[idx]));

        hipStreamAddCallback(stream[idx],CallBack,(void *)i,0);
        
        hipEventRecord(g_stops[i],stream[idx]);

    }
    
    //hipEventRecord(g_stop);
   
    return NULL;/*}}}*/
}


void *cpu_search_up_omp(void *args) {
/*{{{*/

    stick_this_thread_to_core(2);
    struct timeval start;
    struct timeval end;

    int idx = 0;
    const int keys = Para_Search_Bucket / UP_THREAD;
    const int count = (Para_Search_Bucket+keys-1) / keys;
    gettimeofday(&start, NULL); 
    while(idx<total){
        int start = idx*Para_Search_Bucket;
        omp_set_num_threads(UP_THREAD);
        #pragma omp parallel for  
        for(int i=0;i<count;i++){
            stick_this_thread_to_core(i%UP_THREAD+2);
            vector<Inner_node *> nodes(keys,root);
            int relist_idx[keys];
            for(int step = 1; step <=up_height; step++){
                for(int k = 0; k<keys &&  i * keys + k < Para_Search_Bucket; k++){
                   relist_idx[k] = nodes[k]->getChildIdx_avx2(NULL, host_keys[start + i * keys + k]);
                   nodes[k] = static_cast<Inner_node *>(((Inner_node *)nodes[k])->child[relist_idx[k]]);
                   __builtin_prefetch(nodes[k],0,3);
                }
            }
            for(int k=0;k<keys && i*keys+k<Para_Search_Bucket; k++) host_inter_result[start+i*keys+k] = nodes[k]-inner_start;
        }
        key_status++;
        idx++;
    }
    gettimeofday(&end, NULL);
    cout<<"CPU load balance"<<(end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0<<endl;
    return NULL;
/*}}}*/
}
void *cpu_search_down(void *args){
/*{{{*/
    struct timeval start;
    struct timeval end;
    struct timeval start1;
    struct timeval end1;
    double total_time = 0;
    double compute_time = 0;
    int threadID = (long long)args; 
 //   if (threadID==0)
    stick_this_thread_to_core((threadID+2+UP_THREAD)%56);

    const int keys = Para_Search_Bucket/ DOWN_THREAD;// every thread process 16 key for once;
    const int count = (Para_Search_Bucket+keys-1) / keys;//how many batch in bucket
    const int total_keys = (count+DOWN_THREAD-1) / DOWN_THREAD * keys;// keys count  for one thread
    value_t val[total_keys];

    gettimeofday(&start, NULL); 
    
    
    for (int i=0;i<total;i++) {
        int idx = i%2;
        while(stream_flag[idx]<i);
        int start = i * Para_Search_Bucket;

        int process_idx = start+threadID*keys;
        int j = 0;

        gettimeofday(&start1, NULL); 
        while(process_idx<start + Para_Search_Bucket){
            int k = process_idx;
            for(;k < process_idx+keys && k < start+Para_Search_Bucket;k++){ 
                key_t key = host_keys[start + host_pos[k]];
                BLeaf_node *blfnode = bptree->getLeafByIdx(h_gresult[k].blfnode);
                val[j++] = blfnode->findKey(h_gresult[k].relist_idx, key);
            }
            process_idx +=  DOWN_THREAD * keys;
        }
        gettimeofday(&end1, NULL); 
        compute_time += (end1.tv_sec - start1.tv_sec) + (end1.tv_usec-start1.tv_usec) / 1000000.0;  
               //test
   
        /*
        process_idx = start+threadID*keys;
        j = 0;
        while(process_idx<Para_Search_Bucket){
            int k = process_idx;
            for(;k<process_idx+keys && k<Para_Search_Bucket;k++){ 
                key_t key = host_keys[start + host_pos[k]];
                 cout<<key<<" "<<val[j++]<<endl;
            }
            process_idx +=  DOWN_THREAD * keys;
        }
        */

    
    
    }
 //   if(threadID == 0)
    gettimeofday(&end, NULL);
    total_time += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;  
    cpu_thread_total_time[threadID] = total_time;
    cpu_thread_compute_time[threadID] = compute_time;
    return 0;/*}}}*/
}

/*--------------------------------------------------------------------
*               
*               PPI_BPT_v6
*               double buffer
*               using 2 thread
*               batch
*               key sort first, pos back. 
*               whole 
*               CPU multi-thread 
*               balance
*               up_thread and down_thread fixed
*               up omp
*---------------------------------------------------------------------------*/




void PPI_BPT_Search_GPU_V6_balance(BPlusTree &tree,ifstream &search_file){
/*{{{*/
    stick_this_thread_to_core(0);
    hipEventCreate(&g_start);
    hipEventCreate(&g_stop);
    

    bptree = &tree;
    prepareGPU_v2(tree, d_innode, d_prefix);

    rootIdx = tree.getRootIdx();
    root = (Inner_node *)tree.getRoot();
    inner_node_size_wo_last_inner_node = tree.getInnerSize_wo_last_inner_node();
    inner_start = tree.getInnerNodeStart();

    up_height = UP_HEIGHT;
    kernel_height = tree.getHeight()-UP_HEIGHT;
    assert(rootIdx != -1);
    
    //host_malloc
    CUDA_ERROR_HANDLER(hipHostMalloc(&host_keys,sizeof(key_t)*Keys_Count));
   
    //init key 
    int nums = 0;
    string s;
    while(getline(search_file,s)) {
        key_t key;
        sscanf(s.c_str(),TYPE_D,&key);
        host_keys[nums++] = key;
    } 
    total = nums / Para_Search_Bucket;
    
    CUDA_ERROR_HANDLER(hipHostMalloc(&h_gresult,sizeof(GPU_Result)*nums));
    CUDA_ERROR_HANDLER(hipHostMalloc(&host_pos,sizeof(int)*nums));
    CUDA_ERROR_HANDLER(hipHostMalloc(&host_inter_result,sizeof(int)*nums));

    //init key pos
    for (int i=0;i<Para_Search_Bucket;i++) {
        host_pos[i] = i;
        host_pos[i+Para_Search_Bucket] = i;
    }
    for(int i=0;i<2;i++) hipStreamCreate(&stream[i]);

    //gpu_malloc
    CUDA_ERROR_HANDLER(hipMalloc(&d_keys, batch_d_key_size*2));

    CUDA_ERROR_HANDLER(hipMalloc(&d_keys_after, batch_d_key_size*2));
  
    CUDA_ERROR_HANDLER(hipMalloc(&d_pos, batch_pos_size*2));

    CUDA_ERROR_HANDLER(hipMalloc(&d_pos_after, batch_pos_size*2));

    
    CUDA_ERROR_HANDLER(hipMalloc(&d_gresult, batch_gresult_size *2));
    CUDA_ERROR_HANDLER(hipMalloc(&d_inter_result, batch_inter_size *2));

        
    value_t val[Para_Search_Bucket];

    struct timeval start;
    struct timeval end;
    struct timeval start1;
    struct timeval end1;
    double total_time=0;
    double cpu_time = 0;

    CUDA_ERROR_HANDLER(hipMemcpy(d_pos, host_pos, batch_pos_size*2, hipMemcpyHostToDevice));
    
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_keys_after, d_pos,d_pos_after,Para_Search_Bucket,16,32);
    CUDA_ERROR_HANDLER(hipMalloc(&d_temp_storage, temp_storage_bytes*2));



    pthread_t tid;
    if((pthread_create(&tid,NULL,launch_kernel_thread_new_tree_measure_balance,NULL))!=0){
        cout<<"can't create thread\n"<<endl;
    }
   
    if((pthread_create(&tid,NULL,cpu_search_up_omp,NULL))!=0){
        cout<<"can't create thread\n"<<endl;
    }

   vector<pthread_t> tid_arr;
   for(int i=0;i< DOWN_THREAD;i++){
       if((pthread_create(&tid,NULL,cpu_search_down,(void *)i))!=0){
           cout<<"can't create thread\n"<<endl;
       }else{
           tid_arr.push_back(tid);
       }
   }

   
   
   gettimeofday(&start, NULL); 
   for(int i= 0;i<DOWN_THREAD;i++){
       pthread_join(tid_arr[i],NULL);
   }
   gettimeofday(&end, NULL);
   total_time += (end.tv_sec - start.tv_sec) + (end.tv_usec-start.tv_usec) / 1000000.0;  

   // 1
    hipEventSynchronize(g_stop);
    hipEventElapsedTime(&time_gpu,g_start,g_stop);
    
   /* 2
    for (int i=0;i<total;i+=2) {
        float time1 = 0;
        hipEventSynchronize(g_stops[i]);
        hipEventElapsedTime(&time1,g_starts[i], g_stops[i]);
        time_gpu += time1; 
    }
    */
   
   
   
   cout<<"GPU PPI V6 [sort key first,2threads,doublebuffer, whole, new_tree, balance]"<<total * Para_Search_Bucket<<endl;
    cout<<"GPU search total num:"<<total * Para_Search_Bucket<<endl;
    cout<<"GPU time(one stream)     "<<time_gpu/1000<<endl;
    cout<<"total_time:              "<<total_time<<endl;
    //compute time  
    double tmp = 0;
    for(auto t:cpu_thread_compute_time){
        tmp+=t;
    }
    cout<<"average thread compute time:"<<tmp/ DOWN_THREAD<<endl;
    tmp = 0;
    for(auto t:cpu_thread_total_time){
        tmp+=t;
    }
    cout<<"average thread total_time "<<tmp/ DOWN_THREAD<<endl;
   
    CUDA_ERROR_HANDLER(hipHostFree(h_gresult));
    CUDA_ERROR_HANDLER(hipHostFree(host_keys));
    CUDA_ERROR_HANDLER(hipHostFree(host_pos));
    CUDA_ERROR_HANDLER(hipFree(d_innode));
    CUDA_ERROR_HANDLER(hipFree(d_keys));
    CUDA_ERROR_HANDLER(hipFree(d_keys_after));
    CUDA_ERROR_HANDLER(hipFree(d_temp_storage));
    CUDA_ERROR_HANDLER(hipFree(d_gresult));

/*}}}*/

}


