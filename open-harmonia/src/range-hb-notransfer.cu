#include "hip/hip_runtime.h"
#define haha
#ifdef haha

#include<hip/hip_runtime.h>
#include<iostream>
#include<sys/time.h>
#include <assert.h>

#include <string>
#include"ppi-bpt.h"
#include"cuda_utils.h"
#include"mempool.h"
#include"buffer.h"

#include <fstream>
#include "hipcub/hipcub.hpp"
#include "conf.h"

#define M 1000000
#define Keys_Count 80*M

#define  GPU_SWP_SIZE 8 
//#define  GPU_SWP_SIZE (8*Thread_Per_Query) 


#define Block_Dim 16
#define Grid_Dim (8192*SM)


#define Thread_Per_Query 8
//#define Thread_Per_Query (DEFAULT_ORDER/Narrow_Rate) 
#define Sort_Per_Thread (GPU_SWP_SIZE / Thread_Per_Query)


#define Range_Size 8


typedef struct{
    key_t keys[Range_Size];
    value_t vals[Range_Size];
}GPU_Result;


using namespace std;
namespace AAA2{
/*{{{*/
    
    int Para_Search_Bucket = (Grid_Dim * Block_Dim) / Thread_Per_Query * GPU_SWP_SIZE;//851968

    dim3 block_dim(Block_Dim);
    dim3 grid_dim(Grid_Dim);

    int batch_d_key_size = sizeof(key_t) * Para_Search_Bucket;
    long batch_gresult_size = sizeof(GPU_Result) * Para_Search_Bucket;

    key_t **host_keys_begin;
    key_t **host_keys_end;
    GPU_Result  **h_gresult;
    
    key_t **d_keys_begin;
    key_t **d_keys_end;
    
    

    GPU_Result **d_gresult;
    
    

    int rootIdx; 

    int kernel_height;
    int total;
    int total_per_gpu;

    Inner_node **d_innode;
    BLeaf_node **d_bleafnode;
    
    hipStream_t **stream;

    float time_gpu = 0;

    //hipEvent_t g_start,g_stop;
    hipEvent_t *g_start;
    hipEvent_t *g_stop;

    int ngpus;


/*}}}*/
}

using namespace AAA2;
__global__ void range_hb_notransfer_kernel(Inner_node *d_innode, BLeaf_node *d_bleafnode, int root_idx,  key_t *d_keys_begin, key_t *d_keys_end, int tree_height, GPU_Result *d_gresult, int key_count){
    /*{{{*/ 
    int key_base = Block_Dim / Thread_Per_Query * GPU_SWP_SIZE;
    int key_idx = key_base * blockIdx.x + threadIdx.x/ Thread_Per_Query;  

    if(key_idx>=key_count) return;

    int search_idx = threadIdx.x % Thread_Per_Query;
    int query_idx_in_local_block = threadIdx.x / Thread_Per_Query;
    int r = query_idx_in_local_block;//just for simple

    const int row = Block_Dim / Thread_Per_Query;//blockDim.x/ Thread_Per_Query;
    const int row_swp = row * GPU_SWP_SIZE;
    
    __shared__ char flag[row][9];
    __shared__ int start_idx[row_swp];
    __shared__ int relist_idx[row_swp];

    __shared__ int inner_index_result[row];
    //__shared__ char nexthalf[row];

    int stride = Block_Dim / Thread_Per_Query;
 
    for (int k = 0; k<Sort_Per_Thread; k++){
        start_idx[threadIdx.x + k* blockDim.x] = root_idx;
    }

    flag[r][search_idx] = 0;
    flag[r][search_idx+1] = 0;
    char selfFlag;
    
    __syncthreads();

    key_t target;
    
    //without leaf
    for(int i=1;i<tree_height;i++){
        for(int j=0;j<GPU_SWP_SIZE;j++){
            // get key
            __syncthreads();

            int cur_key = key_idx+stride *j;
            if(cur_key>=key_count)continue;
            int cur_r = r+stride *j;

            target = d_keys_begin[cur_key];

            int pos = start_idx[cur_r];
            Inner_node *node = d_innode + pos;
            //search index;
            key_t idx_key = node->inner_index[search_idx];
             
            if (target < idx_key) {
                flag[r][search_idx+1] = 1;
                selfFlag = 1;
            }
            
            __syncthreads();


            if (selfFlag == 1 && flag[r][search_idx] == 0){
                inner_index_result[r] = search_idx;     
            }
             
            selfFlag = 0;
            flag[r][search_idx+1] = 0;
            __syncthreads();

            //------------------------------------------------
            int begin = inner_index_result[r]*8;
            int idx = begin + search_idx;
            key_t key = node->inner_key[idx];

        
            //==== shit 
            if (search_idx==0) {
                if (i == tree_height-1) {
                    start_idx[cur_r] = (int)(node->child[0]); //for leaf   
                    relist_idx[cur_r] = begin + 8;
                }
                else 
                    start_idx[cur_r] = (int)(node->child[begin+8]);
            }

            if (target < key) {
                flag[r][search_idx + 1] = 1;
                selfFlag = 1;
            }

            __syncthreads();
            

            if (selfFlag == 1 && flag[r][search_idx] == 0) {
                if(i==tree_height-1){
                    //start_idx[cur_r] = (BLeaf_node *)node->child[0]; //for leaf   
                    relist_idx[cur_r] = idx;
                }else 
                    start_idx[cur_r] = (int)node->child[idx];
                
            }
            inner_index_result[r] = 0;
            selfFlag = 0;
            flag[r][search_idx+1]=0;
            __syncthreads();
   
        }
    }

    //range
    // 每一个thread处理Sort_Per_Thread个数据。
    //Sort_Per_Thread前面有用过，代表每一个线程应该处理多少数据。
    
    key_idx = Sort_Per_Thread * blockDim.x * blockIdx.x + threadIdx.x;//当前线程应该从哪一个数据开始处理
    r = threadIdx.x; //当前线程第一个处理的数据在Local block中的编号，用于__shared__。
    stride = blockDim.x;    //同一个线程处理的相邻两个数据相差开的位置。 
    for (int j=0;j<Sort_Per_Thread;j++) {
        
        int cur_key = key_idx + stride * j;
        if (cur_key >= key_count) continue;
        
        
        int cur_r = r + stride * j;
        key_t start = d_keys_begin[cur_key]; 
        key_t end = d_keys_end[cur_key];
        GPU_Result &result=d_gresult[cur_key];
        
        int bleaf_pos = start_idx[cur_r];
        char relist_id = relist_idx[cur_r];
        char idx = 0;
       
        BLeaf_node *node = d_bleafnode + bleaf_pos;
        int used = node->used_relist_slot_num;

        bool start_flag = 0;     // when key is larger than start, flag = 1.
        int ans_num = 0;

        while (ans_num < Range_Size) {
            key_t key = node->relist[relist_id].r[idx].r_key; 

            if ( key == Max_Key ) {
                goto next_position2;
            }



            if (start_flag == 0) {
                if (key < start) goto next_position;
                start_flag = 1;
            }
            if (key > end) break;
            
            result.keys[ans_num] = key;
            result.vals[ans_num] = node->relist[relist_id].r[idx].val;
            ans_num++;
            
next_position:
            idx++;
            if (idx == L_Fanout) {
next_position2:
                idx = 0;
                relist_id++;
                if (relist_id  >= used ) {
                    relist_id = 0;
                    bleaf_pos = (int)(node->next);          // node's next; BLeaf_node is sorted.
                    if (bleaf_pos == -1) break;
                    node = d_bleafnode + bleaf_pos;
                    used = node->used_relist_slot_num;
                }
            }
        }
    
    }

    
    /*}}}*/
}




/*--------------------------------------------------------------------------
*
*       range-hb 
*       range-hb(-8thread)-search 
*       range-1thread-scan 
*       one thread process (GPU_SWP_SIZE/Thread_Per_Query) data
*       no transfer
*
*
*-----------------------------------------------------------------------------*/

static void* launch_kernel_thread(void *args){
/*{{{*/
    
    g_start = (hipEvent_t *)malloc(ngpus * sizeof(hipEvent_t));
    g_stop = (hipEvent_t *)malloc(ngpus * sizeof(hipEvent_t));
   



    for (int i=0; i<total_per_gpu; i++) {
        for (int iii=0; iii<ngpus; iii++) {
            hipSetDevice(iii+GPU_START);

            int idx = i%2;

            key_t *tmp_dkeys_begin = d_keys_begin[iii] + i * Para_Search_Bucket;
            key_t *tmp_dkeys_end = d_keys_end[iii] + i * Para_Search_Bucket;


            key_t *tmp_hkeys_begin = host_keys_begin[iii] + i * Para_Search_Bucket; 
            key_t *tmp_hkeys_end = host_keys_end[iii] + i * Para_Search_Bucket; 
            
            CUDA_ERROR_HANDLER(hipMemcpyAsync(tmp_dkeys_begin, tmp_hkeys_begin, batch_d_key_size, hipMemcpyHostToDevice,stream[iii][idx]));
            CUDA_ERROR_HANDLER(hipMemcpyAsync(tmp_dkeys_end, tmp_hkeys_end, batch_d_key_size, hipMemcpyHostToDevice,stream[iii][idx]));
           

        }
    }
   
    for (int iii=0;iii<ngpus;iii++) {
        hipSetDevice(iii+GPU_START);
        hipEventCreate(&(g_start[iii]));
        hipEventCreate(&(g_stop[iii]));
        hipEventRecord(g_start[iii]);
    }
 
     
    
    for(int i=0;i<total_per_gpu;i++){
        for (int iii=0;iii<ngpus;iii++) {
            hipSetDevice(iii+GPU_START);
            
            int idx = i%2;
            int stride = idx*Para_Search_Bucket;

            key_t *tmp_dkeys_begin = d_keys_begin[iii] + i * Para_Search_Bucket;
            key_t *tmp_dkeys_end = d_keys_end[iii] + i * Para_Search_Bucket;
            
            GPU_Result *tmp_d_gresult = d_gresult[iii] + stride; 
            GPU_Result *tmp_h_gresult = h_gresult[iii] + i * Para_Search_Bucket;
        

            
            range_hb_notransfer_kernel<<<grid_dim, block_dim,0,stream[iii][idx]>>>(d_innode[iii],d_bleafnode[iii], rootIdx, tmp_dkeys_begin, tmp_dkeys_end, kernel_height , tmp_d_gresult, Para_Search_Bucket);
           

            /* 
            for (int j=0;j<Range_Size;j++) {
                int stride2 = Para_Search_Bucket / Range_Size; 
                CUDA_ERROR_HANDLER(hipMemcpyAsync(tmp_h_gresult+stride2*j, tmp_d_gresult+stride2*j, batch_gresult_size/Range_Size, hipMemcpyDeviceToHost,stream[iii][idx]));
            }
            //这里是不支持一次性cpy回来这么大的空间，就分成了多次传回来，Range_Size在这里没有什么特别的意义，用多少都行。
            */
        
        }
    }
        
        
        
    for (int iii=0;iii<ngpus;iii++) {
        hipSetDevice(iii+GPU_START);
        hipEventRecord(g_stop[iii]);
    }
    return NULL;/*}}}*/
}

void range_hb_notransfer(BPlusTree &tree,ifstream &search_file){
/*{{{*/

    //hipGetDeviceCount(&ngpus);
    ngpus  = GPU_NUM;
    d_innode = (Inner_node **)malloc(ngpus * sizeof(Inner_node*));
    d_bleafnode = (BLeaf_node **)malloc(ngpus * sizeof(BLeaf_node*));
    
    for (int i=0; i<ngpus;i++) {
        hipSetDevice(i+GPU_START);
        d_innode[i] = prepareGPU_v1(tree);
        prepareGPU_leaf2(tree, d_bleafnode[i]);
    }

    rootIdx = tree.getRootIdx();
    kernel_height = tree.getHeight();
    
    
    assert(rootIdx != -1);
   
    //for balance 
    //host_malloc
    host_keys_begin = (key_t **)malloc(ngpus * sizeof(key_t *));
    host_keys_end = (key_t **)malloc(ngpus * sizeof(key_t *));
    int nKeys_Count = Keys_Count / ngpus;
    for (int i=0;i<ngpus;i++) {
        hipSetDevice(i+GPU_START);
        CUDA_ERROR_HANDLER(hipHostMalloc(&(host_keys_begin[i]),sizeof(key_t)*nKeys_Count));
        CUDA_ERROR_HANDLER(hipHostMalloc(&(host_keys_end[i]),sizeof(key_t)*nKeys_Count));
    }
   
    //init key 
    int nums = 0;
    string s;
    while(getline(search_file,s)) {
        key_t key;
        sscanf(s.c_str(),TYPE_D,&key);
        host_keys_begin[nums/nKeys_Count][nums%nKeys_Count] = key;
        getline(search_file,s);
        sscanf(s.c_str(),TYPE_D,&key);
        host_keys_end[nums/nKeys_Count][nums%nKeys_Count] = Max_Key;
        nums++;
    } 
    total = nums / Para_Search_Bucket;
    total_per_gpu = nKeys_Count / Para_Search_Bucket;
  // 这里使用nKeys_Count(而不是nums/ngpus)是为了使得在nums不是Keys_Count大小时依然在结果上正确。因为任务不是按照保证每个GPU做同量分配的，而是保证前面的gpu可以占满



    //cout<<Para_Search_Bucket<<endl;
    //cout<<nums<<endl;

    h_gresult = (GPU_Result **)malloc(ngpus * sizeof(GPU_Result*));
    for (int i=0;i<ngpus;i++) {
        hipSetDevice(i+GPU_START);
        CUDA_ERROR_HANDLER(hipHostMalloc(&(h_gresult[i]),long(sizeof(GPU_Result))*nKeys_Count));
    }



    stream = (hipStream_t **)malloc(ngpus * sizeof(hipStream_t*));
    d_keys_begin = (key_t **)malloc(ngpus * sizeof(key_t*));
    d_keys_end = (key_t **)malloc(ngpus * sizeof(key_t*));
    
    d_gresult = (GPU_Result **)malloc(ngpus * sizeof(GPU_Result*));
    for (int iii=0; iii<ngpus; iii++) {
        
        hipSetDevice(iii+GPU_START);
        
        stream[iii] = (hipStream_t *)malloc(2 * sizeof(hipStream_t));
        for(int i=0;i<2;i++) hipStreamCreate(&(stream[iii][i]));
    
    
    
        //gpu_malloc
        CUDA_ERROR_HANDLER(hipMalloc(&(d_keys_begin[iii]), batch_d_key_size*total_per_gpu));
        CUDA_ERROR_HANDLER(hipMalloc(&(d_keys_end[iii]), batch_d_key_size*total_per_gpu));
        CUDA_ERROR_HANDLER(hipMalloc(&(d_gresult[iii]), batch_gresult_size *2));
    }


    pthread_t tid;
    if((pthread_create(&tid,NULL,launch_kernel_thread,NULL))!=0){
        cout<<"can't create thread\n"<<endl;
    }
    

    cout<<"range: hb (8thread-search &)  1thread-scan"<<endl;
    cout<<"GPU search total num:"<<ngpus * total_per_gpu * Para_Search_Bucket<<endl;

    pthread_join(tid, NULL);
    for (int i=0;i<ngpus;i++) {
        hipSetDevice(i+GPU_START);
        hipEventSynchronize(g_stop[i]);
          
        CUDA_ERROR_HANDLER(hipEventElapsedTime(&time_gpu,g_start[i],g_stop[i]));
        cout<<"GPU time(one stream):     "<<time_gpu/1000<<endl;
    }

#ifdef ENABLE_TEST

    for (int iii=0;iii<ngpus;iii++) {
        for (int i=0;i<total_per_gpu;i++) {
            int base = i*Para_Search_Bucket;
            for (int j=0; j<Para_Search_Bucket; j++) {
                int idx = base + j;
                key_t key_begin = host_keys_begin[iii][idx];
                key_t key_end = host_keys_end[iii][idx]; 

                cout<<"start: "<<key_begin<<" end: "<<key_end<<endl;
                
                for (int jj = 0; jj<Range_Size;jj++) 
                    cout<<h_gresult[iii][idx].keys[jj]<<" : "<<h_gresult[iii][idx].vals[jj]<<endl;

            }
        }
    
    
    }
#endif

    for (int i=0;i<ngpus;i++) {
        
        hipSetDevice(i+GPU_START);
        CUDA_ERROR_HANDLER(hipHostFree(h_gresult[i]));
        CUDA_ERROR_HANDLER(hipHostFree(host_keys_begin[i]));
        CUDA_ERROR_HANDLER(hipHostFree(host_keys_end[i]));
        CUDA_ERROR_HANDLER(hipFree(d_innode[i]));
        CUDA_ERROR_HANDLER(hipFree(d_bleafnode[i]));
        CUDA_ERROR_HANDLER(hipFree(d_keys_begin[i]));
        CUDA_ERROR_HANDLER(hipFree(d_keys_end[i]));
        CUDA_ERROR_HANDLER(hipFree(d_gresult[i]));
    }

/*}}}*/

}
#endif
